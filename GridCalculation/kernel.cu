#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void memTransfer(int* input) {
  int gID = blockIdx.x * blockDim.x + threadIdx.x;
  printf("tid = %d, gid = %d, value = %d\n", threadIdx.x, gID, input[gID]);
}

int main() {

  int size = 128;
  int byteSize = size * sizeof(int);

  int *sInput;
  sInput = (int *)malloc(byteSize);

  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i < size; ++i) {
    sInput[i] = (int)(rand() & 0xff);
  }

  int *dInput;
  hipMalloc((void **)&dInput, byteSize);

  hipMemcpy(dInput, sInput, byteSize, hipMemcpyHostToDevice);

  dim3 block(64);
  dim3 grid(2);

  memTransfer<<<grid, block>>>(dInput);
  hipDeviceSynchronize();

  hipFree(dInput);
  free(sInput);

  hipDeviceReset();
  return 0;
}